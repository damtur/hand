#include "hip/hip_runtime.h"
#include "GpuFunctions.h"
#include <cutil.h>

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// host code
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gpuBinaryErrode(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations, const int morphType) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		//unsigned char p = 255;
		int iterationKw = iterations * iterations;

		for(int i = -iterations; i <= iterations; ++i){
			for(int j = -iterations; j <= iterations; ++j){
				if((morphType != STRUCT_TYPE_CIRCLE || i*i + j*j < iterationKw) && x + i > 0 && x + i < src.cols && y + j > 0 && y + j < src.rows){
					if(src.ptr(y+j)[x+i] != 255){
						dst.ptr(y)[x] = 0;
						return;
					}
					//p &= src.ptr(y+j)[x+i];
				}
			}
		}
		//dst.ptr(y)[x] = p;
		dst.ptr(y)[x] = 255;
	}
}





__global__ void gpuFastBinaryErrodeStep1(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		for(int i = -iterations; i <= iterations; ++i){
			if(x + i > 0 && x + i < src.cols){
				if(src.ptr(y)[x+i] != 255){
					dst.ptr(y)[x] = 0;
					return;
				}
			}
		}
		dst.ptr(y)[x] = 255;
	}
}
__global__ void gpuFastBinaryErrodeStep2(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		for(int i = -iterations; i <= iterations; ++i){
			if(y + i > 0 && y + i < src.rows){
				if(src.ptr(y+i)[x] != 255){
					dst.ptr(y)[x] = 0;
					return;
				}
			}
		}
		dst.ptr(y)[x] = 255;
	}
}








__global__ void gpuGrayscaleErrode(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations, const int morphType) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		unsigned char p = 255;
		int iterationKw = iterations * iterations;

		for(int i = -iterations; i <= iterations; ++i){
			for(int j = -iterations; j <= iterations; ++j){
				if((morphType != STRUCT_TYPE_CIRCLE || i*i + j*j < iterationKw) && x + i > 0 && x + i < src.cols && y + j > 0 && y + j < src.rows){
					p = (src.ptr(y+j)[x+i] < p) ? src.ptr(y+j)[x+i] : p;
				}
			}
		}
		dst.ptr(y)[x] = p;
	}
}

__global__ void gpuBinaryDilate(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations, const int morphType) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		if(src.ptr(y)[x] == 0 ){
			//unsigned char p = 0;
			int iterationKw = iterations * iterations;

			for(int i = -iterations; i <= iterations; ++i){
				for(int j = -iterations; j <= iterations; ++j){
					if((morphType != STRUCT_TYPE_CIRCLE || i*i + j*j < iterationKw) && x + i > 0 && x + i < src.cols && y + j > 0 && y + j < src.rows){
						//p |= src.ptr(y+j)[x+i];
						if(src.ptr(y+j)[x+i] == 255){
							dst.ptr(y)[x] = 255;
							return;
						}
					}
				}
			}
			dst.ptr(y)[x] = 0;
			//dst.ptr(y)[x] = p;
		}else{
			dst.ptr(y)[x] = 255;
		}
	}
}

__global__ void gpuFastBinaryDilateStep1(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		if(src.ptr(y)[x] == 0 ){

			for(int i = -iterations; i <= iterations; ++i){
				if( x + i > 0 && x + i < src.cols){
					if(src.ptr(y)[x+i] == 255){
						dst.ptr(y)[x] = 255;
						return;
					}
				}
			}
			dst.ptr(y)[x] = 0;
		}else{
			dst.ptr(y)[x] = 255;
		}
	}
}

__global__ void gpuFastBinaryDilateStep2(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		if(src.ptr(y)[x] == 0 ){
			for(int i = -iterations; i <= iterations; ++i){
				if( y + i > 0 && y + i < src.rows){
					if(src.ptr(y+i)[x] == 255){
						dst.ptr(y)[x] = 255;
						return;
					}
				}
			}
			dst.ptr(y)[x] = 0;
		}else{
			dst.ptr(y)[x] = 255;
		}
	}
}





__global__ void gpuGrayscaleDilate(const cv::gpu::DevMem2D_<unsigned char> src, cv::gpu::DevMem2D_<unsigned char> dst, const int iterations, const int morphType) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < src.cols && y < src.rows){
		unsigned char p = 0;
		int iterationKw = iterations * iterations;

		for(int i = -iterations; i <= iterations; ++i){
			for(int j = -iterations; j <= iterations; ++j){
				if((morphType != STRUCT_TYPE_CIRCLE || i*i + j*j < iterationKw) && x + i > 0 && x + i < src.cols && y + j > 0 && y + j < src.rows){
					p = (src.ptr(y+j)[x+i] > p) ? src.ptr(y+j)[x+i] : p;
				}
			}
		}
		dst.ptr(y)[x] = p;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//// client code
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void GpuFunctions::binaryErrode(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	if(morphType == STRUCT_TYPE_RECT){
		gpuFastBinaryErrodeStep1<<<grid, block, 0, gpuStream>>>(src, dst, iterations);
		gpuFastBinaryErrodeStep2<<<grid, block, 0, gpuStream>>>(src, dst, iterations);
	}else{
		gpuBinaryErrode<<<grid, block, 0, gpuStream>>>(src, dst, iterations, morphType);
	}

	
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::grayscaleErrode(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	gpuGrayscaleErrode<<<grid, block, 0, gpuStream>>>(src, dst, iterations, morphType);
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::binaryDilate(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);

	if(morphType == STRUCT_TYPE_RECT){
		gpuFastBinaryDilateStep1<<<grid, block, 0, gpuStream>>>(src, dst, iterations);
		gpuFastBinaryDilateStep2<<<grid, block, 0, gpuStream>>>(src, dst, iterations);
	}else{
		gpuBinaryDilate<<<grid, block, 0, gpuStream>>>(src, dst, iterations, morphType);
	}

	
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::grayscaleDilate(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	gpuGrayscaleDilate<<<grid, block, 0, gpuStream>>>(src, dst, iterations, morphType);
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::binaryOpen(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, cv::gpu::DevMem2D_<unsigned char>& buffor,  const int iterations,  hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);

	if(morphType == STRUCT_TYPE_RECT){
		gpuFastBinaryErrodeStep1<<<grid, block, 0, gpuStream>>>(src, buffor, iterations);
		gpuFastBinaryErrodeStep2<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations);

		gpuFastBinaryDilateStep1<<<grid, block, 0, gpuStream>>>(dst, buffor, iterations);
		gpuFastBinaryDilateStep2<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations);
	}else{
		gpuBinaryErrode<<<grid, block, 0, gpuStream>>>(src, buffor, iterations, morphType);
		gpuBinaryDilate<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations, morphType);
	}

	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::grayscaleOpen(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, cv::gpu::DevMem2D_<unsigned char>& buffor,  const int iterations,  hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	gpuGrayscaleErrode<<<grid, block, 0, gpuStream>>>(src, buffor, iterations, morphType);
	gpuGrayscaleDilate<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations, morphType);	
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::binaryClose(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, cv::gpu::DevMem2D_<unsigned char>& buffor, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);

	if(morphType == STRUCT_TYPE_RECT){
		gpuFastBinaryDilateStep1<<<grid, block, 0, gpuStream>>>(src, buffor, iterations);
		gpuFastBinaryDilateStep2<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations);

		gpuFastBinaryErrodeStep1<<<grid, block, 0, gpuStream>>>(dst, buffor, iterations);
		gpuFastBinaryErrodeStep2<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations);
	}else{
		gpuBinaryDilate<<<grid, block, 0, gpuStream>>>(src, buffor, iterations, morphType);
		gpuBinaryErrode<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations, morphType);
	}
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuFunctions::grayscaleClose(const cv::gpu::DevMem2D_<unsigned char>& src, cv::gpu::DevMem2D_<unsigned char>& dst, cv::gpu::DevMem2D_<unsigned char>& buffor, const int iterations, hipStream_t gpuStream, const int morphType){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	gpuGrayscaleDilate<<<grid, block, 0, gpuStream>>>(src, buffor, iterations, morphType);
	gpuGrayscaleErrode<<<grid, block, 0, gpuStream>>>(buffor, dst, iterations, morphType);
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}