#include "hip/hip_runtime.h"
#include "GpuSkinDetector.h"


#include <cutil.h>

#ifndef MAX
#define MAX(a,b)            (((a) > (b)) ? (a) : (b))
#endif

#ifndef MAXI
#define MAXI(a,b,c)			(((a) > (b)) ? (((a) > (c)) ? (a) : (c)) : (((b) > (c)) ? (b) : (c)))
#endif

#ifndef MIN
#define MIN(a,b)            (((a) < (b)) ? (a) : (b))
#endif

#ifndef MINI
#define MINI(a,b,c)			(((a) < (b)) ? (((a) < (c)) ? (a) : (c)) : (((b) < (c)) ? (b) : (c)))
#endif


/////////////////////////////////////////// EMPTY //////////////////////////////////////////////////////////////////

__global__ void gpuEmptyFunction(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		if(px.r != 0 && px.g != 0 && px.b != 0){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

/////////////////////////////////////////// yCbCr //////////////////////////////////////////////////////////////////

static const int range_cb_min = 110;
static const int range_cb_max = 141;
static const int range_cr_min = 128;//130
static const int range_cr_max= 155;

#ifndef FIT_RANGE
#define FIT_RANGE(v,l,h) (MIN(h, MAX(l, v)))
#endif

__global__ void gpuYCbCrFunction(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;

		//return true;
		// convert to YCbCr 
		//y  =  0.2989f*r + 0.5866f*g + 0.1145f*b + 0.5f;
		float cb = -0.1687f*r - 0.3312f*g + 0.500f*b  + 128.0f;
		float cr =  0.500f*r  - 0.4183f*g - 0.0816f*b + 128.0f;

		FIT_RANGE(cb, 0.0, 255.0);
		FIT_RANGE(cr, 0.0, 255.0);
		// end convert 

		float sum = r+g+b;

		if (
			r>95&&g> 40 && b > 20 
			&& MAX(MAX(r,g),b)-MIN(MIN(r,g),b) > 15 
			&& abs(r-g) > 15 && r > g && r > b 
			&& (b/g<1.249)&& b/g > 0.5 &&
			(sum/(3*r)>0.692)&&//0.696
			(0.3333-b/sum>0.029)&&//0.014
			(g/(3*sum)<0.124)

			||	(3*b*r*r)/(sum*sum*sum) >0.110  //0.1276
			&& ((r*b + g*g ) / g*b) > 5000 
			&& sum/(3*r + (r-g)/sum) < 2.7775 //2.7775 lub czasem 1.06
			&& cb >= range_cb_min 
			&& cb <= range_cb_max
			&& cr >= range_cr_min 
			&& cr <= range_cr_max
			){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

/////////////////////////////////////////// GAUSSIAN //////////////////////////////////////////////////////////////////


__global__ void gpuGausianFunction(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float sum = px.r + px.g + px.b;
		
		if ((px.g/px.b - px.r/px.g<=-0.0905)&&
			((px.g*sum)/(px.b*(px.r-px.g))>3.4857)&&
			((sum*sum*sum)/(3*px.g*px.r*px.r)<=7.397)&&
			(sum/(9*px.r)-0.333 > -0.0976)){

			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

/////////////////////////////////////////// HSV //////////////////////////////////////////////////////////////////


__global__ void gpuHsvFunction(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];

		/* convert to HSV */
		float tc = 0;

		float r_float;
		float g_float;
		float b_float;

		float h;
		float s;
		float v;

		r_float = ((float)px.r)/255.0;
		g_float = ((float)px.g)/255.0;
		b_float = ((float)px.b)/255.0;

		v = MAX(px.r,MAX(px.g,px.b));
		s = (v == 0 ? 0 : ((v - MIN(px.r,MIN(px.g,px.b))) / v));

		if (v == r_float){
			h =       (g_float - b_float) * 60 / s;
		} else if (v == g_float) {
			h = 180 + (b_float - r_float) * 60 / s;
		} else { 
			h = 240 + (r_float - g_float) * 60 / s;
		}

		if (h < 0){
			h += 360;
		}

		s *= 255.0;
		v *= 255.0;
		/* end convert */

		int range_h_min;
		int range_h_max;
		int range_s_min;
		int range_s_max;


		range_h_min = 244;
		range_h_max = 267;
		range_s_min = 87;
		range_s_max = 200;

		float sum = px.r+px.b+px.g;
		if (        
			h >= range_h_min && 
			h <= range_h_max &&      
			s >= range_s_min && 
			s <= range_s_max
			&&px.r>95&&px.g> 40 && px.b > 20 
			&& (px.b/px.g<1.249)&& px.b/px.g > 0.5 &&
			(sum/(3*px.r)>tc/0.692)&&
			(0.3333-px.b/sum>0.029)&&
			(px.g/(3*sum)<0.124)

			||	(3*px.b*px.r*px.r)/(sum*sum*sum) >0.110 && 
			((px.r*px.b + px.g*px.g ) / px.g*px.b) > 5000 &&
			sum/(3*px.r + (px.r-px.g)/sum) <2.7775 


			){

			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

/////////////////////////////////////////// SIMPLE //////////////////////////////////////////////////////////////////


__global__ void gpuSimpleFunction(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;
		
		float sum = r+g+b;

		float a1 = r/b;
		float a2 = (r*b)/(sum*sum);
		float a3 = (r*g)/(sum*sum);

		if ( a1 > 1.185 && a2 > 0.107 && a3 > 0.112){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

__global__ void gpuSimpleFunction2(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];

		float r = px.r;
		float g = px.g;
		float b = px.b;
		
		float sum = r+g+b;

		if (
			r > 95 && g > 40 && b > 20 
			&& MAX(MAX(r,g),b) - MIN(MIN(r,g),b) > 15 
			&& abs(r-g) > 15 
			&& r > g 
			&& r > b 
			&& (b/g < 1.249)
			&& b/g > 0.5 
			&& (sum/(3*r) > 0.692) 
			&& (0.3333-b/sum > 0.029)
			&& (g/(3*sum) < 0.124) || (3*b*r*r)/(sum*sum*sum) > 0.110 
			&& ((r*b + g*g ) / g*b) > 5000 
			&& sum/(3*r + (r-g)/sum) <2.7775 //2.7775 lub czasem 1.06
			){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}
__global__ void gpuSimpleFunction3(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;

		float sum = r+g+b;

		if ((g / b - r / b <= -0.0905) && (sum / (3*r) + (r- g) / sum <= 0.9498)){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}
__global__ void gpuSimpleFunction4(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;

		float sum = r+g+b;

		if((b/g < 1.249)
			&& (sum/(3*r) > 0.692)
			&& (0.3333-b/sum > 0.029)
			&& (g/(3*sum) < 0.124)){

			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}
__global__ void gpuSimpleFunction5(const cv::gpu::DevMem2D_<Pixel> src, cv::gpu::DevMem2D_<Pixel> dst) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;

		float sum = r+g+b;

		if(g/b - r/g<=-0.0905
			&& (g*sum)/(b*(r-g)) > 3.4857
			&& (sum*sum*sum)/(3*g*r*r) <= 7.397
			&& sum/(9*r)-0.333 > -0.0976){
			Pixel white;
			white.r = 255;
			white.g = 255;
			white.b = 255;
			dst.ptr(y)[x] = white;
		}else{
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			dst.ptr(y)[x] = black;
		}
	}
}

__global__ void gpuInitialFilter(cv::gpu::DevMem2D_<Pixel> src) { 
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < src.cols && y < src.rows){
		Pixel px = src.ptr(y)[x];
		float r = px.r;
		float g = px.g;
		float b = px.b;

		float sum = r + g + b;
		if( (b > 160 && r < 180 && g < 180) || //Too much blue
			(g > 160 && r < 180 && b < 180) || //Too much green
			(b < 70 && r < 70 && g < 70) || //Too dark
			//(g > 200 && b < 80 && r < 100) || //Green
			(r+g > 400 && b < 170) || // Too much red and gree ( yellow like colour)
			//(g > 110 && b < 90) || //Yellow like also (bylo 150)
			(b/(sum) > .4) || //Too much blue in contrast to others
			(g/(sum) > .4)  //Too much green in contrast to others
			//(r < 102 && g > 100 && b > 110 && g < 140 && b < 160)|| //Ocean
			//(r>240 && g> 230 && b>230)
			//||(r==255&&g==255&&b==255)//eliminate white

			|| (r<240 && g<240 && b<240 && abs(r-g)<10 && abs(g-b)<20 && abs(r-b) < 30 && r>200)//sprawdzic czy 20-30 to nie za duzo
			|| (r<240 && g<240 && b<240 && abs(r-b)<5)
		){
			Pixel black;
			black.r = 0;
			black.g = 0;
			black.b = 0;
			src.ptr(y)[x] = black;
		}
	}
}

void GpuSkinDetector::initialFilter( cv::gpu::DevMem2D_<Pixel>& src, hipStream_t gpuStream){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	gpuInitialFilter<<<grid, block, 0, gpuStream>>>(src);
			
	if( gpuStream == 0 ){
		hipDeviceSynchronize();
	}
}

void GpuSkinDetector::detectSkin( const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, FUNCTION function, hipStream_t gpuStream){
	dim3 block(16, 16);

	int grida = src.cols / block.x + !!(src.cols % block.x);
	int gridb = src.rows / block.y + !!(src.rows % block.y);

    dim3 grid(grida, gridb);
	
	switch(function){
		case EMPTY:{
			gpuEmptyFunction<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case YCBCR:{
			gpuYCbCrFunction<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case HSV:{
			gpuHsvFunction<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case GAUSSIAN:{
			gpuGausianFunction<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case SIMPLE:{
			gpuSimpleFunction<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}

		case SIMPLE2:{
			gpuSimpleFunction2<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case SIMPLE3:{
			gpuSimpleFunction3<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case SIMPLE4:{
			gpuSimpleFunction4<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
		case SIMPLE5 :{
			gpuSimpleFunction5<<<grid, block, 0, gpuStream>>>(src, dst);
			break;
		}
	}
	
	
	//cudaSafeCall( hipGetLastError() );

    if (gpuStream == 0) {
		hipDeviceSynchronize();
	}
}



void GpuSkinDetector::yCbCrFunction(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ){
	detectSkin(src, dst, YCBCR, gpuStream);
}

void GpuSkinDetector::emptyFunction(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, EMPTY, gpuStream);
}

void GpuSkinDetector::gausianFunction(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, GAUSSIAN, gpuStream);
}

void GpuSkinDetector::hsvFunction(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, HSV, gpuStream);
}

void GpuSkinDetector::simpleFunction(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, SIMPLE, gpuStream);
}

void GpuSkinDetector::simpleFunction2(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, SIMPLE2, gpuStream);
}

void GpuSkinDetector::simpleFunction3(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, SIMPLE3, gpuStream);
}

void GpuSkinDetector::simpleFunction4(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, SIMPLE4, gpuStream);
}

void GpuSkinDetector::simpleFunction5(const cv::gpu::DevMem2D_<Pixel>& src, cv::gpu::DevMem2D_<Pixel>& dst, hipStream_t gpuStream ) {
	detectSkin(src,dst, SIMPLE5, gpuStream);
}




